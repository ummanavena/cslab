#include "hip/hip_runtime.h"
#include "new.h"
#include<sys/time.h>
#include </usr/local/cuda/include/hip/hip_runtime.h>
#include </usr/local/cuda/include/hip/hip_runtime_api.h>
#include<unistd.h>

#define t (((struct struct_graph *)(graph.extra)))
#define t2 (((struct struct_graph *)(graph1.extra)))
__device__ int   changed =0, hchanged =0,changed2=0;

__global__ void relaxgraph(GGraph graph,int x) {
        int id = blockIdx.x * blockDim.x + threadIdx.x+x;
        if (id <graph.npoints){
        if (t->updated[id] == true){
        t->updated[id]=false;
        int ind0 = graph.index[id];
        int ind1 = graph.index[id+1]-graph.index[id];
        for (int ind2 = 0; ind2 < ind1; ind2++) {
                int ut0 = 2 * (ind0 + ind2); //edge index
                int ut1 = graph.edges[ut0].ipe; //dest point
                int ut2 = graph.edges[ut0 + 1].ipe;
                GMIN(&t->dist[ut1], t->dist[id] + ut2, changed);
		//changed=1;
		//printf("changed to %d\n",t->dist[ut1]);
        }
        } }
}
__global__ void relaxgraph2(GGraph graph1,int x) {
        int id = blockIdx.x * blockDim.x + threadIdx.x+x;
        if (id <graph1.npoints){
        if (t2->updated[id] == true){
        t2->updated[id]=false;
        int ind0 = graph1.index[id];
        int ind1 = graph1.index[id+1]-graph1.index[id];
        for (int ind2 = 0; ind2 < ind1; ind2++) {
                int ut0 = 2 * (ind0 + ind2); //edge index
                int ut1 = graph1.edges[ut0].ipe; //dest point
                int ut2 = graph1.edges[ut0 + 1].ipe;
                GMIN(&t2->dist[ut1], t2->dist[id] + ut2, changed2);
		//printf("changed to %d\n",t2->dist[ut1]);
	
        }
        } }
}


__global__ void   reset ( GGraph  graph,int x )
 {
    int id = blockIdx.x * blockDim.x + threadIdx.x + x;
    if(id<graph.npoints){
    t->dist[id] = 1234567890;
    t->olddist[id] = 1234567890;
    t->updated[id] = false;

    }

 }
 __global__ void   reset2 ( GGraph  graph1,int x )
 {
    int id = blockIdx.x * blockDim.x + threadIdx.x + x;
    if(id<graph1.npoints){
    t2->dist[id] = 1234567890;
    t2->olddist[id] = 1234567890;
    t2->updated[id] = false;

    }

 }
__global__ void   reset1 (GGraph graph,int x )
 {
 int id = blockIdx.x * blockDim.x + threadIdx.x + x;
 if(id<graph.npoints){
 if (t->dist[id] < t->olddist[id]) {
       t->updated[id] = true;
 }
 t->olddist[id] = t->dist[id];
 }
}
__global__ void   reset3 (GGraph graph1,int x )
 {
 int id = blockIdx.x * blockDim.x + threadIdx.x + x;
 if(id<graph1.npoints){
 if (t2->dist[id] < t2->olddist[id]) {
       t2->updated[id] = true;
 }
 t2->olddist[id] = t2->dist[id];
 }
}
void   SSSP ( char    *  name )
 {
	HGraph hgraph ;
	GGraph graph;
	GGraph graph1;
	hgraph.read2(name);
	
	int hosthgraph=1;
	hgraph.extra=(struct struct_graph *)malloc(sizeof(struct struct_graph ));
	


	alloc_extra_hgraph(hgraph,hosthgraph,hgraph.npoints);
	hgraph.cloneGPU(graph,0 );
	int TPB=1024;
	int kb;
	if ((graph.npoints / TPB + 1) > (32 * 1024))
	    kb = (32 * 1024);
	else
	    kb = (graph.npoints / TPB + 1);

	int graphflag=0;
	hipSetDevice(0);
	hipMalloc((void **)(&graph.extra),sizeof(struct struct_graph ));
	struct struct_graph ftemp1;
	if(hipMemcpy(&ftemp1,graph.extra,sizeof(struct struct_graph ),hipMemcpyDeviceToHost)!=hipSuccess)printf("memcpyerror 2");

	if(hipMemcpy(graph.extra,&ftemp1,sizeof(struct struct_graph ),hipMemcpyHostToDevice)!=hipSuccess)printf("memcpyerror 3");
	hipSetDevice(0);
	graphflag=1;
	alloc_extra_graph(graph,graphflag);
	int falcvt1;
	hgraph.cloneGPU(graph1,1);
	int TPB1=1024;

	int kb1;
	if ((graph.npoints / TPB + 1) > (32 * 1024))
	    kb1 = (32 * 1024);
	else
	    kb1 = (graph.npoints / TPB + 1);
	int graph1flag=0;
	hipSetDevice(1);
	hipMalloc((void **)(&graph1.extra),sizeof(struct struct_graph ));
	struct struct_graph ftemp4;
	if(hipMemcpy(&ftemp4,graph1.extra,sizeof(struct struct_graph ),hipMemcpyDeviceToHost)!=hipSuccess)printf("memcpyerror 5");
	
	if(hipMemcpy(graph1.extra,&ftemp4,sizeof(struct struct_graph ),hipMemcpyHostToDevice)!=hipSuccess)printf("memcpyerror 6");
	hipSetDevice(0);
	graph1flag=1;
	alloc_extra_graph1(graph1,graph1flag);
	hipSetDevice(0);
	for(int i=0;i<graph.npoints;i+=kb*TPB){
		reset<<<kb,TPB>>>(graph,i);
	}
	hipDeviceSynchronize();
	hipSetDevice(0);
	hipSetDevice(1);
	for(int i=0;i<graph1.npoints;i+=kb1*TPB1){
		reset2<<<kb1,TPB1>>>(graph1,i);
	}
	hipDeviceSynchronize();
	hipSetDevice(0);
	int f2;
	f2=0;
	struct struct_graph cpy5;
	hipMemcpy(&cpy5,((struct struct_graph *)(graph.extra)),sizeof(struct struct_graph ),hipMemcpyDeviceToHost);
	if(hipMemcpy(&(cpy5.dist[0]),&(f2),sizeof(int ),hipMemcpyHostToDevice)!=hipSuccess)printf("memcpyerror 7");
	int f3;
	f3=0;
	struct struct_graph cpy6;
	hipMemcpy(&cpy6,((struct struct_graph *)(graph1.extra)),sizeof(struct struct_graph ),hipMemcpyDeviceToHost);
	if(hipMemcpy(&(cpy6.dist[0]),&(f3),sizeof(int ),hipMemcpyHostToDevice)!=hipSuccess)printf("memcpyerror 8");

	bool updt=true;
	if (hipMemcpy(&(cpy5.updated[0]), &updt, sizeof(bool), hipMemcpyHostToDevice) != hipSuccess)
	    printf("memcpyerror 1");
	if (hipMemcpy(&(cpy6.updated[0]), &updt, sizeof(bool), hipMemcpyHostToDevice) != hipSuccess)
	    printf("memcpyerror 1");


	#pragma omp parallel sections
	{
		#pragma omp sections
		{
		while(1) {
			int f4;
			f4=0;
			//struct struct_graph cpy7;
			
			//hipMemcpy(&cpy7,((struct struct_graph *)(graph1.extra)),sizeof(struct struct_graph ),hipMemcpyDeviceToHost);
			if(hipMemcpyToSymbol(HIP_SYMBOL(changed2),&f4,sizeof(int ),0,hipMemcpyHostToDevice)!=hipSuccess)printf("memcpyerror 11");
			hipSetDevice(1);
			
			for(int i=0;i<graph1.npoints/2;i+=kb1*TPB1){
				//printf("enter 2");
				relaxgraph2<<<kb1,TPB1>>>(graph1,i);
			}
			hipDeviceSynchronize();
			hipSetDevice(0);
			int f5;
			if(hipMemcpyFromSymbol(&f5,HIP_SYMBOL((changed2)),sizeof(int ),0,hipMemcpyDeviceToHost)!=hipSuccess)printf("memcpyerror 13");
			if( f5==0 ){printf("yes2");break;}
			hipSetDevice(0);
	    		for (int pointIdx = 0; pointIdx < graph1.npoints; pointIdx += kb1 * TPB1) {
				reset3<<<kb1, TPB1>>>(graph1, pointIdx);
	    		}
			}
		}
		#pragma omp sections
		{
		while(1) {
			int f6;
			f6=0;
			
			if(hipMemcpyToSymbol(HIP_SYMBOL(changed),&(f6),sizeof(int ),0,hipMemcpyHostToDevice)!=hipSuccess)printf("memcpyerror 11");
			hipSetDevice(0);
			for(int i=0;i<graph.npoints;i+=kb*TPB){
				//printf("enter");
				relaxgraph<<<kb,TPB>>>(graph,i);
			}
			hipDeviceSynchronize();
			hipSetDevice(0);
			int f7;
			if(hipMemcpyFromSymbol(&f7,HIP_SYMBOL(changed),sizeof(int ),0,hipMemcpyDeviceToHost)!=hipSuccess)printf("memcpyerror 13");
			if( f7==0 ){break;}
			hipSetDevice(0);
	    		for (int pointIdx = 0; pointIdx < graph.npoints; pointIdx += kb * TPB) {
				reset1<<<kb, TPB>>>(graph, pointIdx);
	    		}
		}
		}
	}
	struct struct_graph cpy10;
	int *finalDistanceArray = (int *)malloc(sizeof(int) * graph.npoints);
	hipMemcpy(&cpy10,((struct struct_graph *)(graph.extra)),sizeof(struct struct_graph ),hipMemcpyDeviceToHost);
	if(hipMemcpy(finalDistanceArray,(cpy10.dist),sizeof(int)*graph.npoints,hipMemcpyDeviceToHost)!=hipSuccess)printf("memcpyerror 14");
	struct struct_graph cpy11;
	int *finalDistanceArray1 = (int *)malloc(sizeof(int) * graph1.npoints);
	hipMemcpy(&cpy11,((struct struct_graph *)(graph1.extra)),sizeof(struct struct_graph ),hipMemcpyDeviceToHost);
	if(hipMemcpy(finalDistanceArray1,(cpy11.dist),sizeof(int)*graph.npoints,hipMemcpyDeviceToHost)!=hipSuccess)printf("memcpyerror 15");
	for (int i = 0; i < graph1.npoints; i++)
	    printf("%d \n", min(finalDistanceArray1[i],finalDistanceArray[i]));


	return ;
	}
		
int   main ( int   argc ,char    *  argv [ ] )
 {
if(argc>2)FALC_THREADS=atoi(argv[2]);

 if( argc!=4  )
{
printf("error:-exec -t threads  file");
return 1;
}
SSSP(argv[3]);

}

